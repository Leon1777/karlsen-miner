#include "hip/hip_runtime.h"
#include<stdint.h>
#include <assert.h>
#include "keccak-tiny.c"
#include "xoshiro256starstar.c"
#include "fishhash_cuda_kernel.cuh"


typedef uint8_t Hash[32];

typedef union _uint256_t {
    uint64_t number[4];
    uint8_t hash[32];
} uint256_t;

#define BLOCKDIM 1024
#define HASH_HEADER_SIZE 72
#define LIGHT_CACHE_NUM_ITEMS 1179641

#define RANDOM_LEAN 0
#define RANDOM_XOSHIRO 1

DEV_INLINE void keccak_in_place(uint8_t* data) {
    SHA3_512((uint2*)data);
}

#define LT_U256(X,Y) (X.number[3] != Y.number[3] ? X.number[3] < Y.number[3] : X.number[2] != Y.number[2] ? X.number[2] < Y.number[2] : X.number[1] != Y.number[1] ? X.number[1] < Y.number[1] : X.number[0] < Y.number[0])

__constant__ uint8_t hash_header[HASH_HEADER_SIZE];
__constant__ uint256_t target;

extern "C" __global__ void generate_full_dataset_gpu(
    int light_cache_num_items, // 1179641
    hash512* light_cache,
    int full_dataset_num_items, // 37748717
    hash1024* full_dataset
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index % 1000000 == 0 && threadIdx.x == 0) {
        printf("[GPU] Generating DAG item %d / %d\n", index, full_dataset_num_items);
    }
    if (index >= full_dataset_num_items) return;

    fishhash_context ctx = {light_cache_num_items, light_cache, full_dataset_num_items, full_dataset};
    full_dataset[index] = calculate_dataset_item_1024(ctx, index);
}

/*
extern "C" __global__ void build_light_cache_gpu(hash512* cache_out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= LIGHT_CACHE_NUM_ITEMS) return;

    __shared__ hash512 prev;

    if (i == 0) {
        uint8_t seed[32] = {
            0xeb, 0x01, 0x63, 0xae, 0xf2, 0xab, 0x1c, 0x5a, 0x66, 0x31, 0x0c, 0x1c, 0x14, 0xd6, 0x0f, 0x42,
            0x55, 0xa9, 0xb3, 0x9b, 0x0e, 0xdf, 0x26, 0x53, 0x98, 0x44, 0xf1, 0x17, 0xad, 0x67, 0x21, 0x19
        };
        memset(prev.bytes, 0, 64);
        memcpy(prev.bytes, seed, 32);
        keccak_in_place(prev.bytes);
        cache_out[0] = prev;
    } else {
        __syncthreads();
        hash512 item = cache_out[i - 1];
        keccak_in_place(item.bytes);
        cache_out[i] = item;
    }
}
*/

extern "C" {

    __global__ void khashv2_kernel(
            const uint64_t nonce_mask, 
            const uint64_t nonce_fixed, 
            const uint64_t nonces_len, 
            uint8_t random_type, 
            void* states, 
            uint64_t *final_nonce,
            hash1024* dataset,
            hash512* cache
            ) {

        // assuming header_len is 72
        /*
        if (threadIdx.x == 0 && blockIdx.x == 0) {
            printf("khashv2_kernel Thread %d, Block %d\n", threadIdx.x, blockIdx.x);
            printHash("The cache[10] is : ", cache[10].bytes, 128);
            printHash("The cache[42] is : ", cache[42].bytes, 128);
            printHash("The dataset[10] is : ", dataset[10].bytes, 128);
            printHash("The dataset[42] is : ", dataset[42].bytes, 128);
            printHash("The dataset[12345] is : ", dataset[12345].bytes, 128);
        }
        */
        

        int nonceId = threadIdx.x + blockIdx.x*blockDim.x;
        if (nonceId < nonces_len) {
            if (nonceId == 0) *final_nonce = 0;
            uint64_t nonce;
            switch (random_type) {
                case RANDOM_LEAN:
                    nonce = ((uint64_t *)states)[0] ^ nonceId;
                    break;
                case RANDOM_XOSHIRO:
                default:
                    nonce = xoshiro256_next(((ulonglong4 *)states) + nonceId);
                    break;
            }
            nonce = (nonce & nonce_mask) | nonce_fixed;
            // header
            uint8_t input[80];
            memcpy(input, hash_header, HASH_HEADER_SIZE);
            // data
            // TODO: check endianity?
            uint256_t hash_;
            memcpy(input +  HASH_HEADER_SIZE, (uint8_t *)(&nonce), 8);
            hashB3(hash_.hash, input, 80);

            /*
            if (threadIdx.x == 0 && blockIdx.x == 0) {
                printHash("hashb3-1 is : ", hash_.hash, 32);
            }
            */
            
           fishhash_context ctx {
                LIGHT_CACHE_NUM_ITEMS,
                cache,
                FULL_DATASET_NUM_ITEMS,
                dataset
            };

            memset(input, 0, 80);
            memcpy(input, hash_.hash, 32);
            hashFish(&ctx, hash_.hash, input);

            /*
            if (threadIdx.x == 0 && blockIdx.x == 0) {
                printHash("hashFish is : ", hash_.hash, 32);
            }
            */

            memset(input, 0, 80);
            memcpy(input, hash_.hash, 32);
            hashB3(hash_.hash, input, 32);

            /*
            if (threadIdx.x == 0 && blockIdx.x == 0) {
                printHash("hashb3-2 is : ", hash_.hash, 32);
            }
            */
            
            
            if (LT_U256(hash_, target)){
                atomicCAS((unsigned long long int*) final_nonce, 0, (unsigned long long int) nonce);
            }
        }
    }

}